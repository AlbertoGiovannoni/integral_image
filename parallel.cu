#include <stdio.h>
#include <hip/hip_runtime.h>
#include "transpose.cu"
#include "scan.cu"
#include "define.h"

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else if (devProp.minor == 9) cores = mp * 128; // ada lovelace
            else printf("Unknown device type\n");
            break;
        case 9: // Hopper
            if (devProp.minor == 0) cores = mp * 128;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

long long int *createImage(long long int *img, int width, int height) {
    for (int i = 0; i < width * height; ++i) {
        img[i] = i + 1;
    }
    return img;
}

__host__ void printValues(long long int *image, int width, int height, const char *label, const long long int longBytes) {
    long long int *h_output = (long long int *) malloc(longBytes);
    hipMemcpy(h_output, image, longBytes, hipMemcpyDeviceToHost);

    printf("%s", label);
    for (int j = 0; j < height; j++) {
        for (int i = 0; i < width; i++) {
            printf("%lld ", h_output[i + j*width]);
        }
        printf("\n");
    }
}

int main() {
    bool printOutput = true;
    const long long int longBytes = INPUT_SIZE * INPUT_SIZE * sizeof(long long int);

    // Stampa delle specifiche hardware
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    printf("Device: %s\n", deviceProp.name);
    printf("Cores number: %d\n", getSPcores(deviceProp));

    // Popolo la matrice
    long long int *h_input = (long long int *) malloc(longBytes);
    h_input = createImage(h_input, INPUT_SIZE, INPUT_SIZE);

    // Allocazione memoria su device
    long long int *d_input, *d_outputS, *d_outputT;

    hipMalloc((void **) &d_input, longBytes);
    hipMalloc((void **) &d_outputS, longBytes);
    hipMalloc((void **) &d_outputT, longBytes);

    dim3 dimGrid(INPUT_SIZE / TILE_SIZE_T, INPUT_SIZE / TILE_SIZE_T, 1);
    dim3 dimBlock(BLOCK_SIZE_T, BLOCK_SIZE_T, 1);

    // Copia immagine su device
    hipMemcpy(d_input, h_input, longBytes, hipMemcpyHostToDevice);

    // Vettore somme parziali
    long long int *sum;
    hipMalloc((void **) &sum, longBytes);

    // Tempo esecuzione
    hipEvent_t startTot, stopTot;
    hipEventCreate(&startTot);
    hipEventCreate(&stopTot);

    if(printOutput)
        printValues(d_input, INPUT_SIZE, INPUT_SIZE, "Immagine di input: \n", longBytes);



    hipEventRecord(startTot);
    // Scan 1
    scan(d_input, d_outputS, sum, INPUT_SIZE, BLOCK_SIZE_S);
    // Transpose 1
    transpose<<<dimGrid, dimBlock>>>(d_outputS, d_outputT);
    // Scan 2
    scan(d_outputT, d_outputS, sum, INPUT_SIZE, BLOCK_SIZE_S);
    // Transpose 2
    transpose<<<dimGrid, dimBlock>>>(d_outputS, d_outputT);

    hipEventRecord(stopTot);
    hipEventSynchronize(stopTot);



    float millisecondsTot = 0;
    hipEventElapsedTime(&millisecondsTot, startTot, stopTot);

    if(printOutput)
        printValues(d_outputT, INPUT_SIZE, INPUT_SIZE, "Immagine integrale: \n", longBytes);

    printf("\nTotal time: %f milliseconds\n", millisecondsTot);

    // Deallocazione della memoria
    free(h_input);
    hipFree(d_input);
    hipFree(d_outputS);
    hipFree(d_outputT);
    hipFree(sum);

    return 0;
}

