#include "hip/hip_runtime.h"
/*
#include <stdio.h>

#define BLOCK_SIZE 16
#define INPUT_SIZE 16


__global__ void scan(int *input, int inputSize, int *output) {
    __shared__ int tmp[2 * BLOCK_SIZE];

    int row = blockIdx.x;
    int idx = threadIdx.x;
    int offset = row * inputSize;

    tmp[2 * idx] = (2 * idx < inputSize) ? input[2 * idx] : 0;
    tmp[2 * idx + 1] = (2 * idx + 1 < inputSize) ? input[2 * idx + 1] : 0;

    // Up Sweep
    for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
        __syncthreads();
        int index = (idx + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE)
            tmp[index] += tmp[index - stride];
    }

    // Down Sweep
    for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (idx + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE) {
            tmp[index + stride] += tmp[index];
        }
    }
    __syncthreads();

    if (idx < inputSize)
        output[idx] = tmp[idx];
}

int main() {
    const int inputSize = INPUT_SIZE;
    const int arrayBytes = inputSize * sizeof(int);

    int *h_input = (int *) malloc(arrayBytes);
    // Riempo input
    for (int i=0; i<inputSize; i++){
        h_input[i] = i+1;
    }

    int *d_input, *d_output;

    hipMalloc((void **)&d_input, arrayBytes);
    hipMemcpy(d_input, h_input, arrayBytes, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_output, arrayBytes);

    scan<<<1, BLOCK_SIZE>>>(d_input, inputSize, d_output);

    int *h_output = (int *) malloc(arrayBytes);
    hipMemcpy(h_output, d_output, arrayBytes, hipMemcpyDeviceToHost);

    printf("Scan Result:\n");
    for (int i = 0; i < inputSize; i++) {
        printf("%d ", h_output[i]);
    }
    printf("\n");

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
*/



#include <stdio.h>
#include <hip/hip_runtime.h>
#include "define.h"


__global__ void scanParallel(long long int *d_input, long long int *d_output, int inputSize, long long int *sum) {
    __shared__ long long int tmp[2 * BLOCK_SIZE_S];
    int row = blockIdx.x;
    int idx = threadIdx.x;
    int offset = row * inputSize;

    // todo
    /*tmp[2 * idx] = (idx * 2 < inputSize) ? input[offset + idx * 2] : 0; // offset + idx rappresenta l'indice globale
    tmp[2 * idx + 1] = (idx * 2 + 1 < inputSize) ? input[offset + idx * 2 + 1] : 0;*/

    tmp[2 * idx] = d_input[offset + idx * 2]; // offset + idx rappresenta l'indice globale
    tmp[2 * idx + 1] = d_input[offset + idx * 2 + 1];

    // Up Sweep
    for (unsigned int stride = 1; stride <= BLOCK_SIZE_S; stride *= 2) {
        __syncthreads();
        int index = (idx + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE_S)
            tmp[index] += tmp[index - stride];
    }

    // Down Sweep
    for (unsigned int stride = BLOCK_SIZE_S / 2; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (idx + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE_S) {
            tmp[index + stride] += tmp[index];
        }
    }
    __syncthreads();

    //if (idx < inputSize)
    d_output[offset + idx] = tmp[idx];

    if (idx == BLOCK_SIZE_S - 1) {
        sum[row] = tmp[idx];
    }

}

__global__ void add(long long int *output, int length, long long int *n) {
    int blockID = blockIdx.x;
    //int threadID = threadIdx.x;

    int blockOffset = blockID * length;
    output[blockOffset] += n[blockID];
}

void scan(long long int *d_input, long long int *d_output, long long int *sum,int inputSize, int blockSize){
    const int numBlocks = (inputSize + blockSize - 1) / blockSize;
    printf("%d ", numBlocks);

    for (int i = 0; i < numBlocks; i++) {
        // Calcola l'offset in base al blocco corrente
        int offset = i * blockSize;

        // Esegui la scan sulla porzione corrente del vettore di input
        scanParallel<<<inputSize, blockSize>>>(d_input + offset, d_output + offset, inputSize, sum);
        // Esegui l'add sulla porzione corrente del vettore di output
        if (i < numBlocks - 1)
            add<<<inputSize, blockSize>>>(d_input + blockSize * (i + 1), inputSize, sum);
    }
}

/*int main2() {
    const int inputSize = INPUT_SIZE;
    const int blockSize = BLOCK_SIZE;
    bool printOutput = true;
    //const int inputBytes = inputSize * inputSize * sizeof(int); // matrici quadrate
    const long long int longBytes = inputSize * inputSize * sizeof(long long int);
    const int numBlocks = (inputSize + blockSize - 1) / blockSize;


    long long int *h_input = (long long int *) malloc(longBytes);
    // Popolo la matrice
    int c = 1;
    for (int j = 0; j < inputSize; j++) {
        for (int i = 0; i < inputSize; i++) {
            h_input[j * inputSize + i] = c;
            c++;
        }
    }

    long long int *h_output = (long long int *) malloc(longBytes);
    //scan(h_input, h_output, inputSize, blockSize, printOutput);


    free(h_input);
    free(h_output);
    return 0;
}*/
